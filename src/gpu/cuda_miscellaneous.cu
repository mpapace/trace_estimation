#include <mpi.h>
#include "global_struct.h"

extern "C" void get_device_properties() {
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, g.device_id);
  g.warp_size = devProp.warpSize;
}
